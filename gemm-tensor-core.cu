#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <vector>
#include <string>
#include <algorithm>
#include <numeric>
#include <thrust/sort.h>
#include <thrust/unique.h>
#include <unordered_map>
#include <hip/hip_cooperative_groups.h>
#include <mma.h>
#include <stdlib.h>

using namespace std;

#define checkRuntime(call)  check_runtime(call, #call, __LINE__, __FILE__)

static bool inline check_runtime(hipError_t e, const char* call, int line, const char *file){
    if (e != hipSuccess) {
        fprintf(stderr, "CUDA Runtime error %s # %s, code = %s [ %d ] in file %s:%d\n", call, hipGetErrorString(e), hipGetErrorName(e), e, file, line);
        return false;
    }
    return true;
}

enum class MemoryType : int{
    None = 0,
    GPU  = 1,
    Managed = 2,
    Host = 3
};

template<typename T, MemoryType type=MemoryType::GPU>
class Memory{
public:
    T* ptr()   const{return ptr_;}
    size_t size() const{return size_;}
    size_t bytes() const{return size_ * sizeof(T);}
    MemoryType memtype() const{return type;}

    virtual ~Memory(){
        free_memory();
    }

    void alloc_or_resize_to(size_t size){
        if(capacity_ < size){
            free_memory();

            if constexpr(type == MemoryType::GPU)
                checkRuntime(hipMalloc(&ptr_, size * sizeof(T)));
            else if constexpr(type == MemoryType::Managed)
                checkRuntime(hipMallocManaged(&ptr_, size * sizeof(T)));
            else if constexpr(type == MemoryType::Host)
                checkRuntime(hipHostMalloc(&ptr_, size * sizeof(T)));
            capacity_ = size;
        }
        size_ = size;
    }

    void free_memory(){
        if(ptr_){
            if constexpr(type == MemoryType::GPU)
                checkRuntime(hipFree(ptr_));
            else if constexpr(type == MemoryType::Managed)
                checkRuntime(hipFree(ptr_));
            else if constexpr(type == MemoryType::Host)
                checkRuntime(hipHostFree(ptr_));

            ptr_     = nullptr;
            capacity_ = 0;
            size_     = 0;
        }
    }

    bool empty() const{ return ptr_ == nullptr;}

private:
    T* ptr_          = nullptr;
    size_t size_     = 0;
    size_t capacity_ = 0;
};

enum class DataType : int{
    None    = 0,
    Int32   = 1,
    Float16 = 2,
    Float32 = 3
};

struct Tensor{
    void* data = nullptr;
    bool owner = true;
    vector<int64_t> shape;
    DataType dtype;
    bool device = true;
};

static void free_tensor(Tensor* tensor){
    if(tensor){
        if(tensor->data && tensor->owner) {
            if(tensor->device){
                checkRuntime(hipFree(tensor->data));
            }else{
                checkRuntime(hipHostFree(tensor->data));
            }
        }
        delete tensor;
    }
}

static Tensor* create_tensor(vector<int64_t> shape, DataType dtype){
    Tensor* output = new Tensor();
    output->owner = true;
    output->shape = shape;
    output->dtype = dtype;

    size_t volumn = std::accumulate(shape.begin(), shape.begin() + shape.size(), 1, std::multiplies<int>());
    size_t dtype_bytes_map[] = {0, 4, 2, 4};
    checkRuntime(hipMalloc(&output->data, volumn * dtype_bytes_map[(int)dtype]));
    return output;
}

static Tensor* reference_tensor(void* data, vector<int64_t> shape, DataType dtype){

    Tensor* output = new Tensor();
    output->data = data;
    output->owner = false;
    output->shape = shape;
    output->dtype = dtype;
    return output;
}

static Tensor* load_tensor(const std::string& file, bool device=true){

    FILE* f = fopen(file.c_str(), "rb");
    if(f == nullptr) return nullptr;

    int head[3];
    fread(head, 1, sizeof(head), f);
    if(head[0] != 0x33ff1101){
        printf("This is invalid tensor file %s\n", file.c_str());
        fclose(f);
        return nullptr;
    }

    Tensor* output = new Tensor();
    output->owner = true;
    output->device = device;

    int ndim = head[1];
    int dtype = head[2];
    int dims[16];
    fread(dims, 1, ndim * sizeof(int), f);

    output->shape.resize(ndim);
    std::transform(dims, dims + ndim, output->shape.begin(), [](int x){return x;});

    int volumn = std::accumulate(dims, dims + ndim, 1, std::multiplies<int>());
    DataType dtype_map[] = {DataType::Float32, DataType::Float16, DataType::Int32};
    int dtype_bytes_map[]        = {4, 2, 4};
    size_t bytes = dtype_bytes_map[dtype] * volumn;
    vector<unsigned char> host_data(bytes);

    output->dtype = dtype_map[dtype];
    fread(host_data.data(), 1, bytes, f);
    fclose(f);

    if(device){
        checkRuntime(hipMalloc(&output->data, bytes));
        checkRuntime(hipMemcpy(output->data, host_data.data(), bytes, hipMemcpyHostToDevice));
    }else{
        checkRuntime(hipHostMalloc(&output->data, bytes));
        checkRuntime(hipMemcpy(output->data, host_data.data(), bytes, hipMemcpyHostToHost));
    }
    checkRuntime(hipDeviceSynchronize());
    return output;
}

static bool load_tensor_to(const std::string& file, void* to){

    FILE* f = fopen(file.c_str(), "rb");
    if(f == nullptr) return false;

    int head[3];
    fread(head, 1, sizeof(head), f);
    if(head[0] != 0x33ff1101){
        printf("This is invalid tensor file %s\n", file.c_str());
        fclose(f);
        return false;
    }

    int ndim = head[1];
    int dtype = head[2];
    int dims[16];
    fread(dims, 1, ndim * sizeof(int), f);

    int volumn = std::accumulate(dims, dims + ndim, 1, std::multiplies<int>());
    DataType dtype_map[] = {DataType::Float32, DataType::Float16, DataType::Int32};
    int dtype_bytes_map[]        = {4, 2, 4};
    size_t bytes = dtype_bytes_map[dtype] * volumn;
    vector<unsigned char> host_data(bytes);

    fread(host_data.data(), 1, bytes, f);
    fclose(f);

    checkRuntime(hipMemcpy(to, host_data.data(), bytes, hipMemcpyHostToDevice));
    checkRuntime(hipDeviceSynchronize());
    return true;
}

static bool save_tensor(const Tensor* tensor, const std::string& file, hipStream_t stream){

    FILE* f = fopen(file.c_str(), "wb");
    if(f == nullptr){
        printf("Failed to open %s\n", file.c_str());
        return false;
    }

    std::unordered_map<DataType, int> dtype_map{
        {DataType::Float32, 0},
        {DataType::Float16, 1},
        {DataType::Int32, 2}
    };

    std::unordered_map<DataType, int> sizeof_dtype{
        {DataType::Float32, 4},
        {DataType::Float16, 2},
        {DataType::Int32, 4}
    };

    int head[] = {0x33ff1101, (int)tensor->shape.size(), dtype_map[tensor->dtype]};
    int dims[16];
    int i = 0;
    size_t bytes = 1;
    for(auto dim : tensor->shape){
        dims[i++] = dim;
        bytes *= dim;
    }
    bytes *= sizeof_dtype[tensor->dtype];

    std::vector<char> host_data(bytes);
    checkRuntime(hipMemcpyAsync(host_data.data(), tensor->data, bytes, hipMemcpyDeviceToHost, stream));
    checkRuntime(hipStreamSynchronize(stream));

    fwrite(head, 1, sizeof(head), f);
    fwrite(dims, 1, tensor->shape.size() * sizeof(int), f);
    fwrite(host_data.data(), 1, bytes, f);
    fclose(f);
    return true;
}

#define M    320
#define N    320
#define K    320
#define mtile    16
#define ntile     8
#define ktile     8
#define MBLOCK   32
#define NBLOCK   32
#define KBLOCK   32

__device__ void zload_global_to_shared_memory_16byte(void* dst, const void* src, int real_size){

    unsigned int shmem_ptr = __cvta_generic_to_shared(dst);
    asm volatile("cp.async.cg.shared.global [%0], [%1], 16, %2;" ::"r"(shmem_ptr), "l"(src), "r"(real_size));
}

__global__ void gemm(half* a, half* b, half* c){

    __shared__ half shared_a[MBLOCK][KBLOCK];
    __shared__ half shared_b[KBLOCK][NBLOCK];
    int ithread = threadIdx.x;
    int mblock  = blockIdx.x * MBLOCK;
    int nblock  = blockIdx.y * NBLOCK;
    int accumulator[2][4][2] = {0};

    for(int kblock = 0; kblock < K; kblock += KBLOCK){
        for(int iwarp = 0; iwarp < 4; ++iwarp){
            /*每个warp A = 2个16x8    B = 4个8x8，分别排列为一行*/
            zload_global_to_shared_memory_16byte(
                &shared_a[ithread][iwarp * 8], a + (mblock + ithread) * K + kblock + iwarp * 8, 16
            );
    
            zload_global_to_shared_memory_16byte(
                &shared_b[ithread][iwarp * 8], b + (nblock + ithread) * K + kblock + iwarp * 8, 16
            );
    
            asm volatile ("cp.async.wait_all;");
            __syncthreads();

            unsigned int shmem_ptra = __cvta_generic_to_shared(&shared_a[ithread][iwarp * 8]);
            unsigned int shmem_ptrb = __cvta_generic_to_shared(&shared_b[ithread][iwarp * 8]);
    
            uint4 val_a, val_b;
            asm volatile ("ldmatrix.sync.aligned.x4.m8n8.shared.b16 {%0, %1, %2, %3}, [%4];" : "=r"(val_a.x), "=r"(val_a.y), "=r"(val_a.z), "=r"(val_a.w) : "r"(shmem_ptra));
            asm volatile ("ldmatrix.sync.aligned.x4.m8n8.shared.b16 {%0, %1, %2, %3}, [%4];" : "=r"(val_b.x), "=r"(val_b.y), "=r"(val_b.z), "=r"(val_b.w) : "r"(shmem_ptrb));

            for(int i = 0; i < 2; ++i){
                for(int j = 0; j < 4; ++j){
                    asm volatile(
                        "mma.sync.aligned.m16n8k8.row.col.f16.f16.f16.f16 {%0,%1}, {%2,%3}, {%4}, {%5,%6};"
                        : "=r"(accumulator[i][j][0]), "=r"(accumulator[i][j][1])
                        : "r"(((uint2*)&val_a)[i].x), "r"(((uint2*)&val_a)[i].y),
                        "r"(((uint32_t*)&val_b)[j]),
                        "r"(accumulator[i][j][0]), "r"(accumulator[i][j][1])
                    );
                }
            }
        }
    }

    for(int i = 0; i < 2; ++i){
        for(int j = 0; j < 4; ++j){
            *(uint32_t*)&c[((mblock + i * 16 + ithread / 4) * N + nblock + j * 8 + (ithread % 4) * 2)]     = accumulator[i][j][0];
            *(uint32_t*)&c[((mblock + i * 16 + ithread / 4 + 8) * N + nblock + j * 8 + (ithread % 4) * 2)] = accumulator[i][j][1];
        }
    }
}

void __global__ arange(half* p, int n){
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if(idx >= n) return;

    uint32_t randval = idx ^ 0xFFAAC031;
    randval <<= 8;
    randval ^= (uint32_t)&randval;
    randval <<= 5;
    randval ^= 0x3192FF01;
    randval = randval % 100;
    int sign = randval % 2 == 0 ? -1 : 1;
    p[idx] = sign * (int)randval / 1000.0f; 
}

int main(){

    Memory<half> A, B, C;
    A.alloc_or_resize_to(M * K);
    B.alloc_or_resize_to(K * N);
    C.alloc_or_resize_to(M * N);

    hipStream_t stream;
    hipStreamCreate(&stream);

    arange<<<(M * K + 1023) / 1024, 1024, 0, stream>>>(A.ptr(), M * K);
    arange<<<(K * N + 1023) / 1024, 1024, 0, stream>>>(B.ptr(), K * N);
    arange<<<(M * N + 1023) / 1024, 1024, 0, stream>>>(C.ptr(), M * N);

    dim3 block(32, 1);
    dim3 grid((M + MBLOCK - 1) / MBLOCK, (N + NBLOCK - 1) / NBLOCK);
    gemm<<<grid, block, 0, stream>>>(A.ptr(), B.ptr(), C.ptr());
    hipStreamSynchronize(stream);

    save_tensor(reference_tensor(A.ptr(), {M, K}, DataType::Float16), "A.bin", stream);
    save_tensor(reference_tensor(B.ptr(), {K, N}, DataType::Float16), "B.bin", stream);
    save_tensor(reference_tensor(C.ptr(), {M, N}, DataType::Float16), "C.bin", stream);
    return 0;
}